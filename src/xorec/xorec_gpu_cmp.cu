#include "hip/hip_runtime.h"
#include "xorec_gpu_cmp.cuh"
#include "utils.hpp"

int DEVICE_ID;
int MAX_THREADS_PER_BLOCK; 

__device__ __constant__ int WARP_SIZE;

static bool XOREC_GPU_INIT_CALLED = false;

void xorec_gpu_init() {
  if (XOREC_GPU_INIT_CALLED) return;
  XOREC_GPU_INIT_CALLED = true;

  int device_count;

  hipGetDeviceCount(&device_count);

  if (device_count <= 0) throw_error("No CUDA devices found");

  DEVICE_ID = 0;
  hipError_t err = hipSetDevice(DEVICE_ID);
  if (err != hipSuccess) throw_error("Failed to set device");
  
  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, DEVICE_ID);
  MAX_THREADS_PER_BLOCK = device_prop.maxThreadsPerBlock;
  int warp_size = device_prop.warpSize;

  err = hipMemcpyToSymbol(HIP_SYMBOL(WARP_SIZE), &warp_size, sizeof(int));
  if (err != hipSuccess) throw_error("Failed to copy warp size to constant memory"); 

  std::fill_n(COMPLETE_DATA_BITMAP.begin(), XOREC_MAX_DATA_BLOCKS, 1);
}

XorecResult xorec_gpu_encode(
  const uint8_t *XOREC_RESTRICT data_buffer,
  uint8_t *XOREC_RESTRICT parity_buffer,
  size_t block_size,
  size_t num_data_blocks,
  size_t num_parity_blocks
) {
  if (xorec_check_args(block_size, num_data_blocks, num_parity_blocks) != XorecResult::Success) return XorecResult::InvalidCounts;
  if (block_size % sizeof(CUDA_ATOMIC_XOR_T) != 0) return XorecResult::InvalidSize;

  hipMemset(parity_buffer, 0, block_size * num_parity_blocks);
  xorec_gpu_encode_kernel<<<1, MAX_THREADS_PER_BLOCK>>>(data_buffer, parity_buffer, block_size, num_data_blocks, num_parity_blocks);

  return XorecResult::Success;
}


XorecResult xorec_gpu_decode(
  uint8_t *XOREC_RESTRICT data_buffer,
  const uint8_t *XOREC_RESTRICT parity_buffer,
  size_t block_size,
  size_t num_data_blocks,
  size_t num_parity_blocks,
  const uint8_t *XOREC_RESTRICT block_bitmap   ///< Indexing for parity blocks starts at bit 128, e.g. the j-th parity block is at bit 128 + j, j < 128
) {
  if (!recovery_needed(block_bitmap)) return XorecResult::Success;

  if (xorec_check_args(block_size, num_data_blocks, num_parity_blocks) != XorecResult::Success) return XorecResult::InvalidCounts;
  if (block_size % sizeof(CUDA_ATOMIC_XOR_T) != 0) return XorecResult::InvalidSize;

  if (!is_recoverable(block_bitmap, num_data_blocks, num_parity_blocks)) return XorecResult::DecodeFailure;
  for (uint32_t i = 0; i < num_data_blocks;  ++i) {
    if (block_bitmap[i]) continue;
    uint8_t * XOREC_RESTRICT recover_block = data_buffer + i * block_size;
    const uint8_t * XOREC_RESTRICT parity_block = parity_buffer + (i % num_parity_blocks) * block_size;
    hipMemcpy(recover_block, parity_block, block_size, hipMemcpyDeviceToDevice);
    xorec_gpu_decode_kernel<<<1, MAX_THREADS_PER_BLOCK>>>(recover_block, data_buffer, block_size, num_data_blocks, num_parity_blocks, i, i%num_parity_blocks);
  }

  return XorecResult::Success;
}

__global__ void xorec_gpu_decode_kernel(
  uint8_t * XOREC_RESTRICT recover_block,
  const uint8_t *XOREC_RESTRICT data_buffer,
  size_t block_size,
  size_t num_data_blocks,
  size_t num_parity_blocks,
  uint32_t lost_block_idx,
  uint32_t parity_idx
) {
  unsigned num_warps = blockDim.x / WARP_SIZE;
  unsigned warp_idx = threadIdx.x / WARP_SIZE;

  unsigned block_elems = block_size / sizeof(CUDA_ATOMIC_XOR_T); // number of 64-bit elements in a block
  unsigned thread_idx = threadIdx.x % WARP_SIZE;
  
  CUDA_ATOMIC_XOR_T * XOREC_RESTRICT recover_block_64 = reinterpret_cast<CUDA_ATOMIC_XOR_T*>(recover_block);
  for (unsigned i = parity_idx + warp_idx * num_parity_blocks; i < num_data_blocks; i += num_warps * num_parity_blocks) {
    if (i == lost_block_idx) continue;
    const CUDA_ATOMIC_XOR_T * XOREC_RESTRICT data_block_64 = reinterpret_cast<const CUDA_ATOMIC_XOR_T*>(data_buffer + i * block_size);

    for (uint32_t j = thread_idx; j < block_elems; j += WARP_SIZE) {
      atomicXor(&recover_block_64[j], data_block_64[j]);
    }
  }
}



__global__ void xorec_gpu_encode_kernel(
  const uint8_t * XOREC_RESTRICT data_buffer,
  uint8_t * XOREC_RESTRICT parity_buffer,
  size_t block_size,
  size_t num_data_blocks,
  size_t num_parity_blocks
) {
  unsigned num_warps = blockDim.x / WARP_SIZE;
  unsigned warp_idx = threadIdx.x / WARP_SIZE;

  unsigned block_elems = block_size / sizeof(CUDA_ATOMIC_XOR_T); // number of 64-bit elements in a block
  unsigned thread_idx = threadIdx.x % WARP_SIZE;

  for (unsigned i = warp_idx; i < num_data_blocks; i += num_warps) {
    const CUDA_ATOMIC_XOR_T * XOREC_RESTRICT data_block_64 = reinterpret_cast<const CUDA_ATOMIC_XOR_T*>(data_buffer + i * block_size);
    CUDA_ATOMIC_XOR_T * XOREC_RESTRICT parity_block_64 = reinterpret_cast<CUDA_ATOMIC_XOR_T*>(parity_buffer + (i%num_parity_blocks) * block_size);

    for (unsigned j = thread_idx; j < block_elems; j += WARP_SIZE) {
      atomicXor(&parity_block_64[j], data_block_64[j]);
    }
  }
}

