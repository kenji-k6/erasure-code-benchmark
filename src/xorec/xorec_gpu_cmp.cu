#include "hip/hip_runtime.h"
#include "xorec_gpu_cmp.cuh"
#include "utils.hpp"
#include <iostream>

static bool XOREC_GPU_INIT_CALLED = false;

void xorec_gpu_init(size_t num_data_blocks, int device_id) {
  if (XOREC_GPU_INIT_CALLED) return;

  int device_count;

  hipGetDeviceCount(&device_count);

  if (device_count <= 0) throw_error("No CUDA devices found");

  hipError_t err = hipSetDevice(device_id);
  if (err != hipSuccess) throw_error("Failed to set device");
  
  hipDeviceProp_t device_prop;
  err = hipGetDeviceProperties(&device_prop, device_id);
  if (err != hipSuccess) throw_error("Failed to rertrieve device properties");

  COMPLETE_DATA_BITMAP.resize(num_data_blocks);
  std::fill_n(COMPLETE_DATA_BITMAP.begin(), num_data_blocks, 1);

  XOREC_GPU_INIT_CALLED = true;
}

XorecResult xorec_gpu_encode(
  const uint8_t *XOREC_RESTRICT data_buf,
  uint8_t *XOREC_RESTRICT parity_buf,
  size_t num_chunks,
  size_t block_size,
  size_t chunk_data_blocks,
  size_t chunk_parity_blocks,
  size_t num_gpu_blocks,
  size_t threads_per_block
) {
  if (!XOREC_GPU_INIT_CALLED) throw_error("xorec_gpu_init() must be called before calling xorec_encode()");
  XorecResult err = xorec_check_args(data_buf, parity_buf, block_size, chunk_data_blocks, chunk_parity_blocks);
  if (err != XorecResult::Success) return err;
  if (block_size % sizeof(CUDA_ATOMIC_XOR_T) != 0) return XorecResult::InvalidSize;

  hipMemsetAsync(parity_buf, 0, num_chunks * chunk_parity_blocks * block_size);
  xorec_gpu_xor_kernel<<<num_gpu_blocks, threads_per_block>>>(
    data_buf,
    parity_buf,
    num_chunks,
    block_size,
    chunk_data_blocks,
    chunk_parity_blocks
  );

  return XorecResult::Success;
}



XorecResult xorec_gpu_decode(
  uint8_t* XOREC_RESTRICT data_buf,
  uint8_t* XOREC_RESTRICT parity_buf,
  size_t num_chunks,
  size_t block_size,
  size_t chunk_data_blocks,
  size_t chunk_parity_blocks,
  const uint8_t* XOREC_RESTRICT block_bitmap,
  size_t num_gpu_blocks,
  size_t threads_per_block
) {
  if (!XOREC_GPU_INIT_CALLED) throw_error("xorec_gpu_init() must be called before calling xorec_encode()");

  XorecResult err = xorec_check_args(data_buf, parity_buf, block_size, chunk_data_blocks, chunk_parity_blocks);
  if (err != XorecResult::Success) return err;
  if (block_size % sizeof(CUDA_ATOMIC_XOR_T) != 0) return XorecResult::InvalidSize;
  
  bool recover_required = false;
  for (unsigned c = 0; c < num_chunks; ++c) {
    auto chunk_bitmap = block_bitmap + c * (chunk_data_blocks + chunk_parity_blocks);
    auto chunk_data_buf = data_buf + c * chunk_data_blocks * block_size;
    
    if (require_recovery(chunk_data_blocks, chunk_bitmap)) recover_required = true;
    if (!is_recoverable(chunk_data_blocks, chunk_parity_blocks, chunk_bitmap)) return XorecResult::DecodeFailure;
    // Zero out lost blocks
    for (unsigned i = 0; i < chunk_data_blocks; ++i) {
      if (!chunk_bitmap[i]) hipMemsetAsync(chunk_data_buf + i * block_size, 0, block_size);
    }
  }

  if (!recover_required) return XorecResult::Success;

  xorec_gpu_xor_kernel<<<num_gpu_blocks, threads_per_block>>>(
    data_buf,
    parity_buf,
    num_chunks,
    block_size,
    chunk_data_blocks,
    chunk_parity_blocks
  );

  // copy recovered blocks back to data_buf
  for (unsigned c = 0; c < num_chunks; ++c) {
    auto chunk_bitmap = block_bitmap + c * (chunk_data_blocks + chunk_parity_blocks);
    auto chunk_data_buf = data_buf + c * chunk_data_blocks * block_size;
    auto chunk_parity_buf = parity_buf + c * chunk_parity_blocks * block_size;

    for (unsigned i = 0; i < chunk_data_blocks; ++i) {
      if (!chunk_bitmap[i]) hipMemcpyAsync(chunk_data_buf + i * block_size, chunk_parity_buf + (i % chunk_parity_blocks) * block_size, block_size, hipMemcpyDeviceToDevice);
    }
  }

  return XorecResult::Success;
}



__global__ void xorec_gpu_xor_kernel(
  const uint8_t* XOREC_RESTRICT data_buf,
  uint8_t* XOREC_RESTRICT parity_buf,
  size_t num_chunks,
  size_t block_size,
  size_t chunk_data_blocks,
  size_t chunk_parity_blocks
) {
  unsigned num_threads = blockDim.x * gridDim.x;
  unsigned glbl_thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  unsigned tot_elems = num_chunks * chunk_data_blocks * block_size / sizeof(CUDA_ATOMIC_XOR_T);
  unsigned chunk_elems = chunk_data_blocks * block_size / sizeof(CUDA_ATOMIC_XOR_T);
  unsigned block_elems = block_size / sizeof(CUDA_ATOMIC_XOR_T);

  for (unsigned i = glbl_thread_idx; i < tot_elems; i += num_threads) {
    unsigned chunk_idx = i / chunk_elems;
    unsigned block_idx = (i % chunk_elems) / block_elems;
    unsigned parity_idx = block_idx % chunk_parity_blocks;

    const CUDA_ATOMIC_XOR_T * XOREC_RESTRICT data_block = reinterpret_cast<const CUDA_ATOMIC_XOR_T*>(
      data_buf + (chunk_idx * chunk_data_blocks * block_size) + (block_idx * block_size)
    );
    CUDA_ATOMIC_XOR_T * XOREC_RESTRICT parity_block = reinterpret_cast<CUDA_ATOMIC_XOR_T*>(
      parity_buf + (chunk_idx * chunk_parity_blocks * block_size) + (parity_idx * block_size)
    );

    atomicXor(&parity_block[(i%chunk_elems)%block_elems], data_block[(i%chunk_elems)%block_elems]);
  }
}


